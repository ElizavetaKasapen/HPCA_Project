#include "hip/hip_runtime.h"
#include "decision_tree.h"
#include <cmath>
#include <algorithm>
#include <iostream>
#include <map>
#include <vector>
#include <hip/hip_runtime.h>
#include <>

#define MAX_CLASSES 3 

// CUDA kernel: each thread evaluates one candidate split
__global__ void evaluate_candidates(const double* d_data, const int* d_labels,
    int num_samples, int num_features,
    double* d_results, int* d_candidate_feature,
    double* d_candidate_threshold) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int total_candidates = num_features * num_samples;
    if (idx >= total_candidates) return;

    // Determine candidate: each candidate corresponds to (feature_index, sample_index)
    int feature_index = idx / num_samples;
    int sample_index = idx % num_samples;
    double threshold = d_data[sample_index * num_features + feature_index];

    int left_total = 0;
    int right_total = 0;
    int left_counts[MAX_CLASSES] = { 0 };
    int right_counts[MAX_CLASSES] = { 0 };

    // Evaluate the split for all samples using the given feature
    for (int i = 0; i < num_samples; i++) {
        double value = d_data[i * num_features + feature_index];
        int label = d_labels[i];
        if (value <= threshold) {
            left_total++;
            if (label < MAX_CLASSES)
                left_counts[label]++;
        }
        else {
            right_total++;
            if (label < MAX_CLASSES)
                right_counts[label]++;
        }
    }

    // Compute Gini for left branch
    double gini_left = 1.0;
    if (left_total > 0) {
        for (int c = 0; c < MAX_CLASSES; c++) {
            double p = static_cast<double>(left_counts[c]) / left_total;
            gini_left -= p * p;
        }
    }

    // Compute Gini for right branch
    double gini_right = 1.0;
    if (right_total > 0) {
        for (int c = 0; c < MAX_CLASSES; c++) {
            double p = static_cast<double>(right_counts[c]) / right_total;
            gini_right -= p * p;
        }
    }

    // Compute weighted Gini 
    double weighted_gini = 0.0;
    if (num_samples > 0) {
        weighted_gini = (left_total * gini_left + right_total * gini_right) / num_samples;
    }

    // Write results to global memory
    d_results[idx] = weighted_gini;
    d_candidate_feature[idx] = feature_index;
    d_candidate_threshold[idx] = threshold;
}


DecisionTree::DecisionTree(int max_depth) : max_depth(max_depth), root(nullptr) {}

DecisionTree::~DecisionTree() {
    delete_tree(root);
}

// Recursively delete all nodes in the tree
void DecisionTree::delete_tree(Node* node) {
    if (node) {
        delete_tree(node->left);
        delete_tree(node->right);
        delete node;
    }
}

// Finds the most common class label (used for leaf nodes)
int DecisionTree::most_common_label(const std::vector<int>& labels) {
    std::map<int, int> label_count;
    for (int label : labels) {
        label_count[label]++;
    }
    return std::max_element(label_count.begin(), label_count.end(),
        [](const auto& a, const auto& b) { return a.second < b.second; })->first;
}


// Recursively builds the decision tree using the dataset and CUDA for candidate evaluation
Node* DecisionTree::build_tree(const std::vector<std::vector<double>>& data,
    const std::vector<int>& labels, int depth) {
    int num_samples = data.size();
    int num_features = data[0].size();

    //std::cout << "Depth: " << depth << ", Samples: " << num_samples << std::endl;

    // Check stopping conditions
    if (depth >= max_depth || num_samples <= 2 ||
        std::all_of(labels.begin(), labels.end(), [&](int v) { return v == labels[0]; })) {
        Node* leaf = new Node();
        leaf->value = most_common_label(labels);
        leaf->left = nullptr;
        leaf->right = nullptr;
        //std::cout << "Created leaf node with value: " << leaf->value << std::endl;
        return leaf;
    }

    // Flatten 2D data into a contiguous 1D array for CUDA
    std::vector<double> flat_data(num_samples * num_features);
    for (int i = 0; i < num_samples; i++) {
        for (int j = 0; j < num_features; j++) {
            flat_data[i * num_features + j] = data[i][j];
        }
    }

    // Allocate device memory
    double* d_data, * d_results, * d_candidate_threshold;
    int* d_labels, * d_candidate_feature;
    size_t data_size = sizeof(double) * flat_data.size();
    size_t labels_size = sizeof(int) * num_samples;
    int total_candidates = num_features * num_samples;
    size_t candidates_size = sizeof(double) * total_candidates;
    size_t candidate_feature_size = sizeof(int) * total_candidates;

    hipMalloc(&d_data, data_size);
    hipMalloc(&d_labels, labels_size);
    hipMalloc(&d_results, candidates_size);
    hipMalloc(&d_candidate_feature, candidate_feature_size);
    hipMalloc(&d_candidate_threshold, candidates_size);

    hipMemcpy(d_data, flat_data.data(), data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels.data(), labels_size, hipMemcpyHostToDevice);

    // Launch kernel: one thread per candidate split
    int blockSize = 256;
    int numBlocks = (total_candidates + blockSize - 1) / blockSize;
    evaluate_candidates << <numBlocks, blockSize >> > (d_data, d_labels, num_samples,
        num_features,
        d_results, d_candidate_feature,
        d_candidate_threshold);
    hipDeviceSynchronize();

    // Copy candidate evaluation results back to host
    std::vector<double> h_results(total_candidates);
    std::vector<int> h_candidate_feature(total_candidates);
    std::vector<double> h_candidate_threshold(total_candidates);
    hipMemcpy(h_results.data(), d_results, candidates_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_candidate_feature.data(), d_candidate_feature, candidate_feature_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_candidate_threshold.data(), d_candidate_threshold, candidates_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_data);
    hipFree(d_labels);
    hipFree(d_results);
    hipFree(d_candidate_feature);
    hipFree(d_candidate_threshold);

    // Find the best candidate split on the host
    double best_gini = 1.0;
    int best_feature = -1;
    double best_threshold = 0.0;
    for (int i = 0; i < total_candidates; i++) {
        if (h_results[i] < best_gini) {
            best_gini = h_results[i];
            best_feature = h_candidate_feature[i];
            best_threshold = h_candidate_threshold[i];
        }
    }

    // If no valid split was found, create a leaf node
    if (best_feature == -1) {
        Node* leaf = new Node();
        leaf->value = most_common_label(labels);
        leaf->left = nullptr;
        leaf->right = nullptr;
       // std::cout << "Created fallback leaf node with value: " << leaf->value << std::endl;
        return leaf;
    }

    //std::cout << "Best feature: " << best_feature << ", Best threshold: " << best_threshold << std::endl;

    // Split the data into left and right branches based on the best split
    std::vector<std::vector<double>> left_data, right_data;
    std::vector<int> left_labels, right_labels;
    for (int i = 0; i < num_samples; i++) {
        if (data[i][best_feature] <= best_threshold) {
            left_data.push_back(data[i]);
            left_labels.push_back(labels[i]);
        }
        else {
            right_data.push_back(data[i]);
            right_labels.push_back(labels[i]);
        }
    }

    // Recursively build the tree
    Node* node = new Node();
    node->feature_index = best_feature;
    node->threshold = best_threshold;
    node->left = build_tree(left_data, left_labels, depth + 1);
    node->right = build_tree(right_data, right_labels, depth + 1);

    return node;
}

void DecisionTree::fit(const std::vector<std::vector<double>>& data, const std::vector<int>& labels) {
    std::cout << "Starting to build the decision tree..." << std::endl;
    root = build_tree(data, labels, 0);
    std::cout << "Decision tree built successfully." << std::endl;
}


int DecisionTree::predict(const std::vector<double>& sample) {
    Node* node = root;
    while (node->left || node->right) {
        if (sample[node->feature_index] <= node->threshold) {
            node = node->left;
        }
        else {
            node = node->right;
        }
    }
    return node->value;
}
